// qr_decomp_iter.cu - CUDA adaptation of Givens QR iteration for complex matrix

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define N 6
#define ITER_EACH 10
#define DIV_N 5000
#define range 100.0

__device__ hipFloatComplex A[N][N];
__device__ hipFloatComplex c[N], s[N];

__device__ float complex_norm(hipFloatComplex a) {
  return hipCrealf(a) * hipCrealf(a) + hipCimagf(a) * hipCimagf(a);
}

__device__ void givens_rotation(hipFloatComplex A[N][N], hipFloatComplex c[N],
                                hipFloatComplex s[N], int row_id, int col_id) {
  hipFloatComplex a = A[col_id][col_id];
  hipFloatComplex b = A[row_id][col_id];
  float norm = rsqrtf(complex_norm(a) + complex_norm(b));
  c[col_id] = hipCmulf(hipConjf(a), make_hipFloatComplex(norm, 0));
  s[col_id] = hipCmulf(hipConjf(b), make_hipFloatComplex(norm, 0));
}

__device__ void mul_givens_mat(hipFloatComplex A[N][N], hipFloatComplex c[N],
                               hipFloatComplex s[N], int row_id, int col_id,
                               int dir) {
  hipFloatComplex coef_c = c[col_id];
  hipFloatComplex coef_s = s[col_id];

  if (dir == 0) {
    for (int i = 0; i < N; i++) {
      hipFloatComplex a = A[col_id][i];
      hipFloatComplex b = A[row_id][i];
      A[col_id][i] = hipCaddf(hipCmulf(coef_c, a), hipCmulf(coef_s, b));
      A[row_id][i] =
          hipCsubf(hipCmulf(hipConjf(coef_c), b), hipCmulf(hipConjf(coef_s), a));
    }
  } else {
    for (int i = 0; i < N; i++) {
      hipFloatComplex a = A[i][col_id];
      hipFloatComplex b = A[i][row_id];
      A[i][col_id] =
          hipCaddf(hipCmulf(hipConjf(coef_c), a), hipCmulf(hipConjf(coef_s), b));
      A[i][row_id] = hipCsubf(hipCmulf(coef_c, b), hipCmulf(coef_s, a));
    }
  }
}

__device__ void qr_decomp(hipFloatComplex A[N][N], int lim) {
  hipFloatComplex tmp = A[lim][lim];
  hipFloatComplex c[N], s[N];
  for (int i = 0; i < N; i++) {
    A[i][i] = hipCsubf(A[i][i], tmp);
  }
  for (int row_id = 1; row_id <= lim; row_id++) {
    int col_id = row_id - 1;
    givens_rotation(A, c, s, row_id, col_id);
    mul_givens_mat(A, c, s, row_id, col_id, 0);
  }
  for (int row_id = 1; row_id <= lim; row_id++) {
    int col_id = row_id - 1;
    mul_givens_mat(A, c, s, row_id, col_id, 1);
  }
  for (int i = 0; i < N; i++) {
    A[i][i] = hipCaddf(A[i][i], tmp);
  }
}

__device__ void qr_iteration_kernel(hipFloatComplex A[N][N]) {
  int t = ITER_EACH * (N - 1);
  for (int i = 0; i < t; i++) {
    qr_decomp(A, N - 1 - i / ITER_EACH);
  }
}

void print_matrix(const hipFloatComplex h_A[N][N]) {
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      printf("(%5.2f,%5.2f) ", hipCrealf(h_A[i][j]), hipCimagf(h_A[i][j]));
    }
    printf("\n");
  }
}

__global__ void kernel(hipFloatComplex *diag_results) {
  int i = blockIdx.y * blockDim.y + threadIdx.y - DIV_N + 1;
  int j = blockIdx.x * blockDim.x + threadIdx.x - DIV_N + 1;

  if (i < -DIV_N + 1 || i > DIV_N || j < -DIV_N + 1 || j > DIV_N)
    return;

  int grid_size = 2 * DIV_N + 1;
  int idx_i = i + DIV_N - 1;
  int idx_j = j + DIV_N - 1;
  int global_idx = (idx_i * grid_size + idx_j) * N;

  float t1 = 1.0f * i * range / DIV_N;
  float t2 = 1.0f * j * range / DIV_N;

  hipFloatComplex local_A[N][N];
  memset(local_A, 0, sizeof(local_A));

  for (int k = 1; k < N; k++) {
    local_A[k][k - 1] = make_hipFloatComplex(1.0f, 0.0f);
  }
  local_A[0][1] = make_hipFloatComplex(0, 1);
  local_A[0][2] = make_hipFloatComplex(-1, 0);
  local_A[0][3] = make_hipFloatComplex(0, 1);
  local_A[0][0] = make_hipFloatComplex(-t2, -1);
  local_A[0][5] = make_hipFloatComplex(-1, t1);

  qr_iteration_kernel(local_A);

  for (int d = 0; d < N; d++) {
    diag_results[global_idx + d] = local_A[d][d];
  }
}

int main() {
  dim3 blockDim(16, 16);
  dim3 gridDim((2 * DIV_N + 1 + 15) / 16, (2 * DIV_N + 1 + 15) / 16);

  hipFloatComplex *d_diag_results;
  hipFloatComplex *h_diag_results;

  int total_points = (2 * DIV_N + 1) * (2 * DIV_N + 1);
  hipMalloc(&d_diag_results, sizeof(hipFloatComplex) * total_points * N);
  h_diag_results =
      (hipFloatComplex *)malloc(sizeof(hipFloatComplex) * total_points * N);

  kernel<<<gridDim, blockDim>>>(d_diag_results);
  hipMemcpy(h_diag_results, d_diag_results,
             sizeof(hipFloatComplex) * total_points * N, hipMemcpyDeviceToHost);

  /*
  for (int idx = 100000; idx < 160000; ++idx) {
    printf("Point %d: ", idx);
    for (int d = 0; d < N; ++d) {
      cuFloatComplex val = h_diag_results[idx * N + d];
      printf("(%5.2f, %5.2f) ", cuCrealf(val), cuCimagf(val));
    }
    printf("\n");
  }
  */

  return 0;
}

